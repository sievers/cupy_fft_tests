//nvcc -o libpycufft.so pycufft.cu -shared -lcufft -Xcompiler -fPIC -lgomp

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <omp.h>
#include <hip/hip_complex.h>





extern "C"
{
void get_work_sizes_r2c(long int *sz, int nsize, long int *nbytes)
{
  size_t nb_max=0;
  for (int i=0;i<nsize;i++)
    {
      hipfftHandle plan;
      if (i==0)
	printf("plan size is %ld\n",sizeof(hipfftHandle));
      int n=sz[2*i];
      int ntrans=sz[2*i+1];
      int rank=1; //we're doing 1D transforms
      int nn=(n/2)+1;
      int istride=1;
      int idist=nn;
      int oembed=nn;
      if (hipfftPlanMany(&plan, rank, &n, &nn, istride, idist,&oembed, istride,oembed, HIPFFT_R2C,ntrans)!=HIPFFT_SUCCESS) {
	fprintf(stderr,"Error in planning r2c with dimensions %d %d\n",n,ntrans);
	*nbytes=-1;
	return;
	  
      }
      hipfftSetAutoAllocation(plan,1);
      size_t nb;
      if (hipfftGetSize(plan,&nb)!=HIPFFT_SUCCESS) {
	fprintf(stderr,"Error in querying size wth dimensions %d %d\n",n,ntrans);
	*nbytes=-1;
	return;
      }
      if (nb>nb_max)
	nb_max=nb;
      if (hipfftDestroy(plan)!= HIPFFT_SUCCESS) {
	fprintf(stderr,"Error destroying plan.\n");
	*nbytes=-1;
	return;
      }
    }
}
}
/*--------------------------------------------------------------------------------*/

void cufft_c2r(float *out, hipfftComplex *data, int len, int ntrans, int isodd)
{
  int nout=2*(len-1)-isodd;
  //float *out;
  //hipMalloc(&out,sizeof(float)*nout*ntrans);
  hipfftHandle plan;
  
  if (hipfftPlan1d(&plan,nout,HIPFFT_C2R, ntrans)!=HIPFFT_SUCCESS)
    fprintf(stderr,"Error planning dft\n");
  //hipDeviceSynchronize();
  //double t1=omp_get_wtime();
  if (hipfftExecC2R(plan,data,out)!=HIPFFT_SUCCESS)
    fprintf(stderr,"Error executing dft\n");
  //hipDeviceSynchronize();
  //double t2=omp_get_wtime();
  //printf("took %12.4g seconds to do fft.\n",t2-t1);

  if (hipfftDestroy(plan)!= HIPFFT_SUCCESS)
    fprintf(stderr,"Error destroying plan.\n");
}
/*--------------------------------------------------------------------------------*/
void cufft_c2r_wplan(float *out, hipfftComplex *data, hipfftHandle plan)
{
  if (hipfftExecC2R(plan,data,out)!=HIPFFT_SUCCESS)
    fprintf(stderr,"Error executing idft\n");
  hipDeviceSynchronize();
  
}
/*--------------------------------------------------------------------------------*/
void cufft_c2r_columns(float *out, hipfftComplex *data,int len, int ntrans, int isodd)
{
  int nout=2*(len-1)+isodd;
  hipfftHandle plan;
  int rank=1;
  int inembed[rank] = {ntrans};
  int onembed[rank]={ntrans};
  int istride=ntrans;
  int idist=1;
  int ostride=ntrans;
  int odist=1;
  if (hipfftPlanMany(&plan,rank,&nout,inembed,istride,idist,onembed,ostride,odist,HIPFFT_C2R,ntrans)!=HIPFFT_SUCCESS)
    fprintf(stderr,"Error planning DFT in c2r_columns.\n");
  if (hipfftExecC2R(plan,data,out)!=HIPFFT_SUCCESS)
    fprintf(stderr,"Error executing DFT in c2r_columns.\n");
  if (hipfftDestroy(plan)!=HIPFFT_SUCCESS)
    fprintf(stderr,"Error destroying plan in c2r_columns.\n");

}

/*--------------------------------------------------------------------------------*/
extern "C" {
void cufft_c2r_host(float *out, hipfftComplex *data, int n, int m, int isodd,int axis)
{
  float *dout;
  hipfftComplex *din;
  int nn;
  if (axis==0)
    nn=2*(n-1)+isodd;
  else
    nn=2*(m-1)+isodd;
  if (hipMalloc((void **)&din,sizeof(hipfftComplex)*n*m)!=hipSuccess)
    fprintf(stderr,"error in hipMalloc\n");
  if (hipMemcpy(din,data,n*m*sizeof(hipfftComplex),hipMemcpyHostToDevice)!=hipSuccess)
    fprintf(stderr,"Error copying data to device.\n");
  if (axis==0) {
    if (hipMalloc((void **)&dout,sizeof(float)*nn*m)!=hipSuccess)
      fprintf(stderr,"error in hipMalloc\n");
    cufft_c2r_columns(dout,din,n,m,isodd);
    //printf("copying %d %d\n",nn,m);
    if (hipMemcpy(out,dout,sizeof(float)*nn*m,hipMemcpyDeviceToHost)!=hipSuccess)
      fprintf(stderr,"Error copying result to host in c2r\n");
  }
  else {
    if (hipMalloc((void **)&dout,sizeof(float)*n*nn)!=hipSuccess)
      fprintf(stderr,"error in hipMalloc\n");
    cufft_c2r(dout,din,m,n,isodd);
    //printf("copying %d %d\n",n,nn);
    if (hipMemcpy(out,dout,sizeof(float)*nn*n,hipMemcpyDeviceToHost)!=hipSuccess)
      fprintf(stderr,"Error copying result to host in c2r\n");
  

  }
}
}

/*--------------------------------------------------------------------------------*/
void cufft_r2c(hipfftComplex *out, float *data, int len, int ntrans)
{
  //int nout=len/2+1;
  hipfftHandle plan;
  
  if (hipfftPlan1d(&plan,len,HIPFFT_R2C, ntrans)!=HIPFFT_SUCCESS)
    fprintf(stderr,"Error planning dft\n");
  //hipDeviceSynchronize();
  //double t1=omp_get_wtime();
  if (hipfftExecR2C(plan,data,out)!=HIPFFT_SUCCESS)
    fprintf(stderr,"Error executing dft\n");
  //hipDeviceSynchronize();
  //double t2=omp_get_wtime();
  //printf("r2c took %12.4g\n",t2-t1);

  if (hipfftDestroy(plan)!= HIPFFT_SUCCESS)
    fprintf(stderr,"Error destroying plan.\n");
}
/*--------------------------------------------------------------------------------*/
void cufft_r2c_wplan(hipfftComplex *out, float *data, int len, int ntrans,hipfftHandle plan)
{
  if (hipfftExecR2C(plan,data,out)!=HIPFFT_SUCCESS)
    fprintf(stderr,"Error executing dft\n");
}

/*--------------------------------------------------------------------------------*/
void cufft_r2c_columns(hipfftComplex *out, float *data, int len, int ntrans)
{
  //int nout=len/2+1;
  //printf("performing %d transforms of length %d %d\n",ntrans,len,nout);

  hipfftHandle plan;
  int rank=1;
  int inembed[rank] = {len};
  int onembed[rank]={ntrans};
  int istride=ntrans;
  int idist=1;
  int ostride=ntrans;
  int odist=1;
  //if (hipfftPlanMany(&plan,1,&nout,&one,len,1,&one,nout,1,HIPFFT_R2C,ntrans)!=HIPFFT_SUCCESS)
  //if (hipfftPlanMany(&plan,rank,&len,inembed,len,1,onembed,nout,1,HIPFFT_R2C,ntrans)!=HIPFFT_SUCCESS)
  if (hipfftPlanMany(&plan,rank,&len,inembed,istride,idist,onembed,ostride,odist,HIPFFT_R2C,ntrans)!=HIPFFT_SUCCESS)
    fprintf(stderr,"Error planning DFT in r2c_columns.\n");
  if (hipfftExecR2C(plan,data,out)!=HIPFFT_SUCCESS)
    fprintf(stderr,"Error executing DFT in r2c_columns.\n");
  if (hipfftDestroy(plan)!=HIPFFT_SUCCESS)
    fprintf(stderr,"Error destroying plan in r2c_columns.\n");
  
}






/*--------------------------------------------------------------------------------*/

extern "C" {
void cufft_r2c_gpu(hipfftComplex *out, float *data, int n, int m, int axis)
{
  if (axis==1)
    cufft_r2c(out,data,m,n);
  else
    cufft_r2c_columns(out,data,n,m);
}
}
/*--------------------------------------------------------------------------------*/

extern "C" {
  void cufft_r2c_gpu_wplan(hipfftComplex *out, float *data, int n, int m, int axis,hipfftHandle *plan)
{
  if (axis==1)
    cufft_r2c_wplan(out,data,m,n,*plan);
  else
    cufft_r2c_columns(out,data,n,m);
}
}
/*--------------------------------------------------------------------------------*/

extern "C" {
void cufft_c2r_gpu_wplan(float  *out, hipfftComplex *data, hipfftHandle *plan)
{
  cufft_c2r_wplan(out,data,*plan);
}
}
/*--------------------------------------------------------------------------------*/

extern "C" {
void cufft_c2r_gpu(float *out, hipfftComplex *data, int n, int m, int axis,int isodd)
{
  if (axis==1)
    cufft_c2r(out,data,m,n,isodd);
  else
    cufft_c2r_columns(out,data,n,m,isodd);
}
}
/*--------------------------------------------------------------------------------*/
extern "C" {
void get_plan_size(hipfftHandle *plan, size_t *sz)
{
  if (hipfftGetSize(*plan,sz)!=HIPFFT_SUCCESS)
    fprintf(stderr,"Error querying plan size.\n");
}
}
/*--------------------------------------------------------------------------------*/
extern "C" {
  void get_plan_r2c(int n, int m, int axis,hipfftHandle *plan,int alloc)
{
  if (axis==1) {
    if (hipfftPlan1d(plan,m,HIPFFT_R2C,n)!=HIPFFT_SUCCESS)
      fprintf(stderr,"Error planning dft.\n");
    else {
      size_t sz=0;
      if (hipfftGetSize(*plan,&sz)!=HIPFFT_SUCCESS)
	fprintf(stderr,"Error getting work size.\n");
      //printf("works size is %ld\n",sz);
      if (alloc==0) {
	hipfftSetAutoAllocation(*plan,0);
	void *ptr=NULL;
	hipfftSetWorkArea(*plan,ptr);
      }
    }
  }
      
}
}
	
/*--------------------------------------------------------------------------------*/
extern "C" {
void get_plan_c2r(int n, int m, int axis,hipfftHandle *plan,int alloc)
//make sure n and m correspond to the size of the *output* transform
{
  if (axis==1) {
    if (hipfftPlan1d(plan,m,HIPFFT_C2R,n)!=HIPFFT_SUCCESS)
      fprintf(stderr,"Error planning idft.\n");
    else {
      if (alloc==0) {
	hipfftSetAutoAllocation(*plan,0);
	void *ptr=NULL;
	hipfftSetWorkArea(*plan,ptr);
      }
    }
  }
}
}
	
/*--------------------------------------------------------------------------------*/
extern "C" {
void destroy_plan(hipfftHandle *plan)
{
  if (hipfftDestroy(*plan)!=HIPFFT_SUCCESS)
    fprintf(stderr,"Error destroying plan.\n");
}
}
	
/*--------------------------------------------------------------------------------*/
extern "C" {
void set_plan_scratch(hipfftHandle plan,void *buf)
{
  if (hipfftSetWorkArea(plan,buf)!=HIPFFT_SUCCESS)
    fprintf(stderr,"Error assigning buffer in set_plan_scratch.\n");
  //else
  //printf("successfully assigned buffer.\n");
	  
}
}

/*--------------------------------------------------------------------------------*/
extern "C" {
void cufft_r2c_host(hipfftComplex *out, float *data, int n, int m, int axis)
{
  hipfftComplex *dout;
  float *din;
  int nn;
  if (axis==0)
    nn=n/2+1;
  else
    nn=m/2+1;
  if (hipMalloc((void **)&din,sizeof(float)*n*m)!=hipSuccess)
    fprintf(stderr,"error in hipMalloc\n");
  if (hipMemcpy(din,data,n*m*sizeof(float),hipMemcpyHostToDevice)!=hipSuccess)
    fprintf(stderr,"Error copying data to device.\n");
  if (axis==0) {
    if (hipMalloc((void **)&dout,sizeof(hipfftComplex)*nn*m)!=hipSuccess)
      fprintf(stderr,"error in hipMalloc\n");
    cufft_r2c_columns(dout,din,n,m);
    //printf("copying %d %d\n",nn,m);
    if (hipMemcpy(out,dout,sizeof(hipfftComplex)*nn*m,hipMemcpyDeviceToHost)!=hipSuccess)
      fprintf(stderr,"Error copying result to host in r2c\n");
  }
  else {
    if (hipMalloc((void **)&dout,sizeof(hipfftComplex)*n*nn)!=hipSuccess)
      fprintf(stderr,"error in hipMalloc\n");
    cufft_r2c(dout,din,m,n);
    //printf("copying %d %d\n",n,nn);
    if (hipMemcpy(out,dout,sizeof(hipfftComplex)*nn*n,hipMemcpyDeviceToHost)!=hipSuccess)
      fprintf(stderr,"Error copying result to host in r2c\n");
  
  }
}
}



/*================================================================================*/


#if 0

int main(int argc, char *argv[])
{
  printf("Hello world!\n");
  int ndet=1000;
  int nsamp=1<<18;
  printf("nsamp is %d\n",nsamp);

  float *fdat=(float *)malloc(sizeof(float)*ndet*nsamp);
  if (fdat!=NULL)
    printf("successfully malloced array on host.\n");

  float *ddat;
  if (hipMalloc((void **)&ddat,sizeof(float)*nsamp*ndet)!=hipSuccess)
    fprintf(stderr,"error in hipMalloc\n");
  hipComplex *dtrans;
  if (hipMalloc((void **)&dtrans,sizeof(hipComplex)*nsamp*ndet)!=hipSuccess)
    fprintf(stderr,"error in hipMalloc\n");

  
  
}
#endif
